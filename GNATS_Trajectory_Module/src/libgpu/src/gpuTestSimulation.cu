#include "hip/hip_runtime.h"
/*
------------------------------------- Credits -----------------------------------------------------------------
Generalized National Airspace Trajectory Simulation (GNATS) software
2017-2021 GNATS Development Team at Optimal Synthesis Inc. are:
Team Lead, Software Architecture and Algorithms: Dr. P. K. Menon
Algorithms and Prototyping: Dr. Parikshit Dutta
Java and C++ Code Development: Oliver Chen and Hari N. Iyer
Illustrative Examples in Python and MATLAB: Dr. Parikshit Dutta, Dr. Bong-Jun Yang, Hari Iyer
Illustrative Examples in SciLab and R: Hari Iyer
Acknowledgements: 
GNATS software was developed under the Arizona State University Subaward No. 18-275 under the NASA University Leadership Initiative Prime Contract No. NNX17AJ86A, with Professor Yongming Liu serving as the Principal Investigator. 
Beta Testing outside Optimal Synthesis Inc. was carried out at Arizona State University under the direction of Professor Yongming Liu, at Vanderbilt University under the direction of Professor Sankaran Mahadevan and Professor Pranav Karve, at the Southwest Research Institute under the direction of Dr. Baron Bichon and Dr. Erin DeCarlo, and at Carnegie-Mellon University under the direction of Professor Pingbo Tang.
NASA Technical points-of-contact: Dr. Anupa Bajwa, Dr. Kaushik Datta, Dr. John Cavolowsky, Dr. Kai Goebel
------------------------------------Legacy Source Code--------------------------------------------------------
Legacy Code for the GNATS software was derived from the software packages developed under the following NASA Small Business Innovation Research Projects:
1. 2004-2006 NASA Contract No. NNA05BE64C with Dr. Shon Grabbe of NASA Ames Research Center as the Technical Monitor.
2. 2008-2010 NASA Contract No. NNX08CA02C with Dr. Joseph Rios of Ames Research Center as the Technical Monitor.
3. 2010-2011 NASA Phase III Contract No. NNA10DC12C with Joseph Rios of Ames Research Center as the Technical Monitor.
3. 2016-2018 NASA Contract No. NNX16CL11C with Dr. Nash’at Ahmad of NASA Langley Research Center as the Technical Monitor.
Contributors to these SBIR projects at Optimal Synthesis Inc. were: Dr. P. K. Menon (Principal Investigator), Jason Kwan (Software Engineer), Gerald M. Diaz (Software Engineer), Dr. Monish Tandale (Research Scientist), Dr. Prasenjit Sengupta (Research Scientist), Dr. Sang-Gyun Park (Research Scientist) and Dr. Parikshit Dutta (Research Scientist).
The inspiration for the SBIR projects is derived from the FACET software developed at NASA Ames Research Center by Drs. Banavar Sridhar, Dr. Karl Bilimoria, Dr. Gano Chatterji, Dr. Shon Grabbe and Dr. Kapil Sheth.
---------------------------------------------------------------------------------------------------------------------
*/

#include "gpuTestSimulation.h"

#include "cuda_compat.h"

#include <hiprand/hiprand_kernel.h>
//#include ""
#include "hip/hip_runtime_api.h"
#include "hip/device_functions.h"

#include "CUDA_Aircraft_new.h"
#include "CUDA_Aircraft_new_2.h"

#include"matrixCU.h"

#include <iostream>
#include <cstdio>

typedef struct _waypoint_oliver {
	char* wpname; // = NULL;

	_waypoint_node_t* prev_node_ptr; // = NULL;
	_waypoint_node_t* next_node_ptr; // = NULL;
} waypoint_oliver_t;

using namespace std;

//float* d_latitude_deg;
float* host_latitude_deg;
vector<float> host_longitude_deg;

waypoint_oliver_t* host_waypoint_ptr;





__device__ float* dev_latitude_deg;
//__device__ CUDA_Aircraft_new dev_cudaObj;

__device__ vector<float>* dev_longitude_deg;

__device__ waypoint_oliver_t dev_waypoint;
__device__ waypoint_oliver_t* dev_waypoint_ptr;


TestKernelParameters* host_struct_var;
__device__ TestKernelParameters* dev_struct_ptr;
__device__ TestKernelParameters dev_struct_var;


__device__ Matrix *pR;






__global__ void gpuTest_global_func0(void) {
	printf("global_func0() starting\n");
}






__device__ void gpuTest_global_func1_layer2(float* inputPtr) {
	printf("gpuTest_global_func1_layer2() starting\n");
	inputPtr[0] = 99.99;
}

__global__ void gpuTest_global_func1(float* inputPtr) {
	printf("gpuTest_global_func1() starting\n");

	gpuTest_global_func1_layer2(inputPtr);
	//inputPtr[0] = 33.33;
}




__device__ void gpuTest_global_func2_layer2(waypoint_oliver_t input_param) {
	printf("gpuTest_global_func2_layer2() starting\n");

	//dev_latitude_deg[0] = 44.44;
//printf("gpuTest_global_func2_layer2() --> dev_latitude_deg[0] = %f\n", dev_latitude_deg[0]);
	//printf("gpuTest_global_func2_layer2() --> dev_waypoint_ptr->wpname = %s\n", dev_waypoint_ptr->wpname);

printf("gpuTest_global_func2_layer2() --> dev_waypoint.wpname = %s\n", dev_waypoint.wpname);
	printf("gpuTest_global_func2_layer2() --> input_param.wpname = %s\n", input_param.wpname);

	printf("gpuTest_global_func2_layer2() ending\n");
}

__global__ void gpuTest_global_func2(waypoint_oliver_t input_param) {
	printf("gpuTest_global_func2() starting\n");

	//dev_latitude_deg[0] = 44.44;
	gpuTest_global_func2_layer2(input_param);
}






__global__ void gpuTest_global_func3(CUDA_Aircraft_new cudaObj) {
	printf("gpuTest_global_func3() starting\n");

	//gpuTest_global_func3_layer2(inputPtr);
	cudaObj.latitude_deg[0] = 77.77;
}






__global__ void gpuTest_global_func4(vector<float>* inputPtr) {
	printf("gpuTest_global_func4() starting\n");

	//inputPtr->push_back(33.33);
}





__global__ void gpuTest_global_func5(TestKernelParameters inputPtr) {
printf("gpuTest_global_func5() starting\n");

	//gpuTest_global_func1_layer2(inputPtr);

	//inputPtr->latitude_deg[0] = 33.33;
	inputPtr.latitude_deg[0] = 33.33 + 100 * 25;

printf("gpuTest_global_func5() --> inputPtr.latitude_deg[0] = %f\n", inputPtr.latitude_deg[0]);
	printf("gpuTest_global_func5() ending\n");
}




__device__ void gpuTest_global_func6_layer2(CUDA_Aircraft_new_2* inputPtr) {
	//inputPtr->setValue(0, 33.33);
	//inputPtr->setValue(0, float(33.33));
}

__global__ void gpuTest_global_func6(CUDA_Aircraft_new_2* inputPtr) {
	printf("gpuTest_global_func6() starting\n");

	gpuTest_global_func6_layer2(inputPtr);
	//inputPtr->setValue(0, 33.33);
}






void test1() {
	const int na = 5, nb = 4;
	    float a[na] = { 1.2, 3.4, 5.6, 7.8, 9.0 };
	    float *_a, b[nb];

	    size_t sza = size_t(na) * sizeof(float);
	    size_t szb = size_t(nb) * sizeof(float);

	    hipFree(0);

	    hipMalloc((void **)&_a, sza );
	    hipMemcpy( _a, a, sza, hipMemcpyHostToDevice);
	    hipMemcpy( b, _a+1, szb, hipMemcpyDeviceToHost);

	    for(int i=0; i<nb; i++)
	        printf("test1() --> %d %f\n", i, b[i]);
}




// *********************************************************************************************

// Good

int propagate_flights_gpuTest(const float& input_t_end,
		const float& input_t_step,
		const float& input_t_step_terminal,
		const float& input_t_step_airborne) {
	printf("propagate_flights_gpuTest() starting\n");

//test1();

	host_latitude_deg = (float*)calloc(1, sizeof(float));
printf("propagate_flights_gpuTest() --> stop point 0\n");
	host_latitude_deg[0] = 11.11;
printf("propagate_flights_gpuTest() --> stop point 00\n");
	host_longitude_deg.push_back(11.11);

//d_latitude_deg = host_latitude_deg;


	host_waypoint_ptr = (waypoint_oliver_t*)calloc(1, sizeof(waypoint_oliver_t));
	host_waypoint_ptr->wpname = (char*)calloc(4, sizeof(char));
	strcpy(host_waypoint_ptr->wpname, "ABC");
	host_waypoint_ptr->wpname[3] = '\0';

printf("propagate_flights_gpuTest() --> stop point 1\n");

	// Good
	hipMalloc(&dev_latitude_deg, 1*sizeof(float));
printf("propagate_flights_gpuTest() --> stop point 2\n");

	hipMalloc((void**)&dev_longitude_deg, 1*sizeof(vector<float>));

printf("propagate_flights_gpuTest() --> stop point 3\n");


	hipMalloc((void**)&dev_waypoint_ptr, 1*sizeof(waypoint_oliver_t));
printf("propagate_flights_gpuTest() --> stop point 4 --> strlen = %d\n", strlen(host_waypoint_ptr->wpname));

//hipMalloc((void**)dev_waypoint.wpname, 3 * sizeof(char));
hipMalloc( (void **) &dev_waypoint.wpname, 3*sizeof(char));

printf("propagate_flights_gpuTest() --> stop point 5\n");

//hipMalloc((void**)dev_waypoint_ptr->wpname, 3 * sizeof(char));
printf("propagate_flights_gpuTest() --> stop point 6\n");

// Good
//hipMemcpy(dev_waypoint.wpname, host_waypoint_ptr->wpname, 3 * sizeof(char), hipMemcpyHostToDevice);

// Not working
hipMemcpyToSymbol(HIP_SYMBOL(dev_waypoint.wpname), &(host_waypoint_ptr->wpname), sizeof(char*), 0, hipMemcpyHostToDevice);

printf("propagate_flights_gpuTest() --> stop point 7\n");


//	//hipMemcpyToSymbol(HIP_SYMBOL(dev_latitude_deg), host_latitude_deg, 1 * sizeof(float));
//	//hipMemcpyToSymbol(HIP_SYMBOL(&dev_latitude_deg), &host_latitude_deg, 1 * sizeof(float));
//	hipMemcpy(dev_latitude_deg, host_latitude_deg, sizeof(float), hipMemcpyHostToDevice);
////hipMemcpy(dev_latitude_deg, d_latitude_deg, sizeof(float), hipMemcpyHostToDevice);

	hipMemcpy(dev_longitude_deg, &host_longitude_deg, sizeof(vector<float>), hipMemcpyHostToDevice);



	printf("propagate_flights_gpuTest() --> (BEFORE) host_latitude_deg[0] = %f\n", host_latitude_deg[0]);



	// Call device
	//gpuTest_global_func0 <<<1, 1>>> ();

	// Good
	//gpuTest_global_func1 <<<1, 1>>> (dev_latitude_deg);

	// Good.  Working.
	//gpuTest_global_func4 <<<1, 1>>> (dev_longitude_deg);

	gpuTest_global_func2 <<<1, 1>>> (dev_waypoint);

	hipDeviceSynchronize();




//	//hipMemcpyToSymbol(HIP_SYMBOL(host_latitude_deg), dev_latitude_deg, 1 * sizeof(float));
//	//hipMemcpyToSymbol(HIP_SYMBOL(&host_latitude_deg), &dev_latitude_deg, 1 * sizeof(float));
//hipMemcpy(host_latitude_deg, dev_latitude_deg, sizeof(float), hipMemcpyDeviceToHost);
//	//hipMemcpy(dev_latitude_deg, host_latitude_deg, sizeof(float), hipMemcpyDeviceToHost);
////hipMemcpy(d_latitude_deg, dev_latitude_deg, sizeof(float), hipMemcpyDeviceToHost);

	hipMemcpy(&host_longitude_deg, dev_longitude_deg, sizeof(vector<float>), hipMemcpyDeviceToHost);





	printf("propagate_flights_gpuTest() --> (AFTER) host_latitude_deg[0] = %f\n", host_latitude_deg[0]);
	//printf("propagate_flights_gpuTest() --> (AFTER) dev_latitude_deg[0] = %f\n", dev_latitude_deg[0]);
//printf("propagate_flights_gpuTest() --> (AFTER) d_latitude_deg[0] = %f\n", d_latitude_deg[0]);




	hipFree(dev_latitude_deg);
	hipFree(dev_longitude_deg);

	free(host_latitude_deg);

	printf("propagate_flights_gpuTest() ending\n");

	return 0;
}






/*
int propagate_flights_gpuTest(const float& input_t_end,
		const float& input_t_step,
		const float& input_t_step_terminal,
		const float& input_t_step_airborne) {
	printf("propagate_flights_gpuTest() starting\n");



	host_latitude_deg = (float*)calloc(1, sizeof(float));
	host_latitude_deg[0] = 11.11;

//d_latitude_deg = host_latitude_deg;




	// Good
	//hipMalloc(&dev_latitude_deg, 1*sizeof(float));





	//CUDA_Aircraft_new cudaObj();
	dev_cudaObj = CUDA_Aircraft_new();




//	//hipMemcpyToSymbol(HIP_SYMBOL(dev_latitude_deg), host_latitude_deg, 1 * sizeof(float));
//	//hipMemcpyToSymbol(HIP_SYMBOL(&dev_latitude_deg), &host_latitude_deg, 1 * sizeof(float));
//	hipMemcpy(dev_latitude_deg, host_latitude_deg, sizeof(float), hipMemcpyHostToDevice);
////hipMemcpy(dev_latitude_deg, d_latitude_deg, sizeof(float), hipMemcpyHostToDevice);

	hipMemcpy(dev_cudaObj.latitude_deg, host_latitude_deg, sizeof(float), hipMemcpyHostToDevice);

	printf("propagate_flights_gpuTest() --> (BEFORE) host_latitude_deg[0] = %f\n", host_latitude_deg[0]);



	// Call device
	//gpuTest_global_func0 <<<1, 1>>> ();

	// Good
	//gpuTest_global_func1 <<<1, 1>>> (dev_latitude_deg);

	// Not working
	//gpuTest_global_func2 <<<1, 1>>> ();

	gpuTest_global_func3 <<<1, 1>>> (dev_cudaObj);

	hipDeviceSynchronize();




//	//hipMemcpyToSymbol(HIP_SYMBOL(host_latitude_deg), dev_latitude_deg, 1 * sizeof(float));
//	//hipMemcpyToSymbol(HIP_SYMBOL(&host_latitude_deg), &dev_latitude_deg, 1 * sizeof(float));
//hipMemcpy(host_latitude_deg, dev_latitude_deg, sizeof(float), hipMemcpyDeviceToHost);
//	//hipMemcpy(dev_latitude_deg, host_latitude_deg, sizeof(float), hipMemcpyDeviceToHost);
////hipMemcpy(d_latitude_deg, dev_latitude_deg, sizeof(float), hipMemcpyDeviceToHost);

	hipMemcpy(host_latitude_deg, dev_cudaObj.latitude_deg, sizeof(float), hipMemcpyDeviceToHost);






	printf("propagate_flights_gpuTest() --> (AFTER) host_latitude_deg[0] = %f\n", host_latitude_deg[0]);
	//printf("propagate_flights_gpuTest() --> (AFTER) dev_latitude_deg[0] = %f\n", dev_latitude_deg[0]);
//printf("propagate_flights_gpuTest() --> (AFTER) d_latitude_deg[0] = %f\n", d_latitude_deg[0]);




	hipFree(dev_latitude_deg);

	free(host_latitude_deg);

	printf("propagate_flights_gpuTest() ending\n");

	return 0;
}
*/




// Good - Struct TestKernelParameters
/*
int propagate_flights_gpuTest(const float& input_t_end,
		const float& input_t_step,
		const float& input_t_step_terminal,
		const float& input_t_step_airborne) {
printf("propagate_flights_gpuTest() starting\n");

	// Prepare host variable
	host_struct_var = (TestKernelParameters*)malloc(sizeof(TestKernelParameters));
	host_struct_var->latitude_deg = (float*)calloc(1, sizeof(float));
	host_struct_var->latitude_deg[0] = 12.12;



	// Good
	//hipMalloc(&dev_latitude_deg, 1*sizeof(float));


// Allocate memory
hipMalloc( (void **) &dev_struct_var.latitude_deg, 1*sizeof(float));


	//hipMalloc(&dev_struct_ptr, 1*sizeof(TestKernelParameters));



//hipMemcpy(dev_struct_var.latitude_deg, host_latitude_deg, sizeof(float), hipMemcpyHostToDevice);
// Copy data from host to device
hipMemcpy(dev_struct_var.latitude_deg, host_struct_var->latitude_deg, sizeof(float), hipMemcpyHostToDevice);





	printf("propagate_flights_gpuTest() --> (BEFORE) host_struct_var->latitude_deg[0] = %f\n", host_struct_var->latitude_deg[0]);

	// Call device function
	gpuTest_global_func5 <<<1, 1>>> (dev_struct_var);

	//hipDeviceSynchronize();



// Copy data from device to host
hipMemcpy(host_struct_var->latitude_deg, dev_struct_var.latitude_deg, sizeof(float), hipMemcpyDeviceToHost);


	printf("propagate_flights_gpuTest() --> (AFTER) host_struct_var->latitude_deg[0] = %f\n", host_struct_var->latitude_deg[0]);



	hipFree(dev_latitude_deg);
	hipFree(dev_longitude_deg);

	hipFree(dev_struct_ptr);

	free(host_latitude_deg);

	printf("propagate_flights_gpuTest() ending\n");

	return 0;
}
*/






const int N = 1000;

__global__ void initialize(Matrix *R) {
	int i= blockIdx.x * blockDim.x + threadIdx.x;
	if (i < N) {
		hiprandState state;
		hiprand_init(clock64(), i, 0, &state);
		//R->assignValue2(i, hiprand_uniform(&state));

		double d11 = 11.11;
		R->assignValue2(i, d11);
	}
}



// Using class.  Working good
/*
int propagate_flights_gpuTest(const float& input_t_end,
		const float& input_t_step,
		const float& input_t_step_terminal,
		const float& input_t_step_airborne) {
printf("propagate_flights_gpuTest() starting\n");

	Matrix R(N);

    //Matrix *pR;
    hipMallocManaged(&pR, sizeof(Matrix));
    *pR = R;

    initialize<<<4,256>>>(pR);

    hipDeviceSynchronize();

    pR->displayArray();

	printf("propagate_flights_gpuTest() ending\n");

	return 0;
}
*/




// Test Class.  Not working
/*
int propagate_flights_gpuTest(const float& input_t_end,
		const float& input_t_step,
		const float& input_t_step_terminal,
		const float& input_t_step_airborne) {
	printf("propagate_flights_gpuTest() starting\n");

	host_latitude_deg = (float*)calloc(1, sizeof(float));
	host_latitude_deg[0] = 11.11;


	printf("propagate_flights_gpuTest() --> stop point 1\n");


	// Good
	//hipMalloc(&dev_latitude_deg, 1*sizeof(float));


	cudaObj_2 = CUDA_Aircraft_new_2();

	printf("propagate_flights_gpuTest() --> stop point 2\n");

	CUDA_Aircraft_new_2* cudaPtr;

	hipMallocManaged(&cudaPtr,sizeof(CUDA_Aircraft_new_2));

	*cudaPtr = cudaObj_2;

	cudaPtr->setValue(0, host_latitude_deg[0]);





//	//hipMemcpyToSymbol(HIP_SYMBOL(dev_latitude_deg), host_latitude_deg, 1 * sizeof(float));
//	//hipMemcpyToSymbol(HIP_SYMBOL(&dev_latitude_deg), &host_latitude_deg, 1 * sizeof(float));
//	hipMemcpy(dev_latitude_deg, host_latitude_deg, sizeof(float), hipMemcpyHostToDevice);
////hipMemcpy(dev_latitude_deg, d_latitude_deg, sizeof(float), hipMemcpyHostToDevice);

	hipMemcpy(dev_longitude_deg, host_longitude_deg, sizeof(vector<float>), hipMemcpyHostToDevice);



	printf("propagate_flights_gpuTest() --> (BEFORE) host_latitude_deg[0] = %f\n", host_latitude_deg[0]);



	// Call device

	// Good
	//gpuTest_global_func1 <<<1, 1>>> (dev_latitude_deg);

	gpuTest_global_func6 <<<1, 1>>> (cudaPtr);

	// Not working
	//gpuTest_global_func2 <<<1, 1>>> ();

	hipDeviceSynchronize();




//	//hipMemcpyToSymbol(HIP_SYMBOL(host_latitude_deg), dev_latitude_deg, 1 * sizeof(float));
//	//hipMemcpyToSymbol(HIP_SYMBOL(&host_latitude_deg), &dev_latitude_deg, 1 * sizeof(float));
//hipMemcpy(host_latitude_deg, dev_latitude_deg, sizeof(float), hipMemcpyDeviceToHost);
//	//hipMemcpy(dev_latitude_deg, host_latitude_deg, sizeof(float), hipMemcpyDeviceToHost);
////hipMemcpy(d_latitude_deg, dev_latitude_deg, sizeof(float), hipMemcpyDeviceToHost);

	hipMemcpy(host_longitude_deg, dev_longitude_deg, sizeof(vector<float>), hipMemcpyDeviceToHost);





//	printf("propagate_flights_gpuTest() --> (AFTER) host_latitude_deg[0] = %f\n", host_latitude_deg[0]);
//	//printf("propagate_flights_gpuTest() --> (AFTER) dev_latitude_deg[0] = %f\n", dev_latitude_deg[0]);
////printf("propagate_flights_gpuTest() --> (AFTER) d_latitude_deg[0] = %f\n", d_latitude_deg[0]);

	printf("propagate_flights_gpuTest() --> (AFTER) cudaPtr->getValue(0) = %f\n", cudaPtr->getValue(0));



	hipFree(dev_latitude_deg);
	hipFree(dev_longitude_deg);

	free(host_latitude_deg);

	printf("propagate_flights_gpuTest() ending\n");

	return 0;
}
*/
